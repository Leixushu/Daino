
#include "DAINO.h"
#include "CUFLU.h"

#ifdef GPU



extern real (*d_Flu_Array_F_In )[FLU_NIN ][ FLU_NXT*FLU_NXT*FLU_NXT ];
extern real (*d_Flu_Array_F_Out)[FLU_NOUT][ PS2*PS2*PS2 ];
extern real (*d_Flux_Array)[9][NCOMP][ PS2*PS2 ];
extern real  *d_MinDtInfo_Fluid_Array;

// global memory arrays in different models
#if   ( MODEL == HYDRO )
#if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
extern real (*d_PriVar)     [5][ FLU_NXT*FLU_NXT*FLU_NXT ];
extern real (*d_Slope_PPM_x)[5][ N_SLOPE_PPM*N_SLOPE_PPM*N_SLOPE_PPM ];
extern real (*d_Slope_PPM_y)[5][ N_SLOPE_PPM*N_SLOPE_PPM*N_SLOPE_PPM ];
extern real (*d_Slope_PPM_z)[5][ N_SLOPE_PPM*N_SLOPE_PPM*N_SLOPE_PPM ];
extern real (*d_FC_Var_xL)  [5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ];
extern real (*d_FC_Var_xR)  [5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ];
extern real (*d_FC_Var_yL)  [5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ];
extern real (*d_FC_Var_yR)  [5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ];
extern real (*d_FC_Var_zL)  [5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ];
extern real (*d_FC_Var_zR)  [5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ];
extern real (*d_FC_Flux_x)  [5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ];
extern real (*d_FC_Flux_y)  [5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ];
extern real (*d_FC_Flux_z)  [5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ];
#endif // #if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )

#elif ( MODEL == MHD )
#warning : WAIT MHD !!!

#elif ( MODEL != ELBDM )
#warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#endif // MODEL

extern hipStream_t *Stream;




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_MemFree_Fluid
// Description :  Free GPU and CPU memory previously allocated by the function "CUAPI_MemAllocate_Fluid"
//
// Parameter   :  GPU_NStream : Number of CUDA streams for the asynchronous memory copy
//-------------------------------------------------------------------------------------------------------
void CUAPI_MemFree_Fluid( const int GPU_NStream )
{

// free the device memory (in all models)
   if ( d_Flu_Array_F_In        != NULL )    CUDA_CHECK_ERROR(  hipFree( d_Flu_Array_F_In        )  );
   if ( d_Flu_Array_F_Out       != NULL )    CUDA_CHECK_ERROR(  hipFree( d_Flu_Array_F_Out       )  );
   if ( d_Flux_Array            != NULL )    CUDA_CHECK_ERROR(  hipFree( d_Flux_Array            )  );
   if ( d_MinDtInfo_Fluid_Array != NULL )    CUDA_CHECK_ERROR(  hipFree( d_MinDtInfo_Fluid_Array )  );
   d_Flu_Array_F_In        = NULL;
   d_Flu_Array_F_Out       = NULL;
   d_Flux_Array            = NULL;
   d_MinDtInfo_Fluid_Array = NULL;


// free the device memory (in different models)
#  if   ( MODEL == HYDRO )
#  if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
   if ( d_PriVar      != NULL )    CUDA_CHECK_ERROR(  hipFree( d_PriVar      )  );

   if ( d_Slope_PPM_x != NULL )    CUDA_CHECK_ERROR(  hipFree( d_Slope_PPM_x )  );
   if ( d_Slope_PPM_y != NULL )    CUDA_CHECK_ERROR(  hipFree( d_Slope_PPM_y )  );
   if ( d_Slope_PPM_z != NULL )    CUDA_CHECK_ERROR(  hipFree( d_Slope_PPM_z )  );

   if ( d_FC_Var_xL   != NULL )    CUDA_CHECK_ERROR(  hipFree( d_FC_Var_xL   )  );
   if ( d_FC_Var_xR   != NULL )    CUDA_CHECK_ERROR(  hipFree( d_FC_Var_xR   )  );
   if ( d_FC_Var_yL   != NULL )    CUDA_CHECK_ERROR(  hipFree( d_FC_Var_yL   )  );
   if ( d_FC_Var_yR   != NULL )    CUDA_CHECK_ERROR(  hipFree( d_FC_Var_yR   )  );
   if ( d_FC_Var_zL   != NULL )    CUDA_CHECK_ERROR(  hipFree( d_FC_Var_zL   )  );
   if ( d_FC_Var_zR   != NULL )    CUDA_CHECK_ERROR(  hipFree( d_FC_Var_zR   )  );

   if ( d_FC_Flux_x   != NULL )    CUDA_CHECK_ERROR(  hipFree( d_FC_Flux_x   )  );
   if ( d_FC_Flux_y   != NULL )    CUDA_CHECK_ERROR(  hipFree( d_FC_Flux_y   )  );
   if ( d_FC_Flux_z   != NULL )    CUDA_CHECK_ERROR(  hipFree( d_FC_Flux_z   )  );

   d_PriVar      = NULL;

   d_Slope_PPM_x = NULL;
   d_Slope_PPM_y = NULL;
   d_Slope_PPM_z = NULL;

   d_FC_Var_xL   = NULL;
   d_FC_Var_xR   = NULL;
   d_FC_Var_yL   = NULL;
   d_FC_Var_yR   = NULL;
   d_FC_Var_zL   = NULL;
   d_FC_Var_zR   = NULL;

   d_FC_Flux_x   = NULL;
   d_FC_Flux_y   = NULL;
   d_FC_Flux_z   = NULL;
#  endif // #if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )

#  elif ( MODEL == MHD )
#  warning : WAIT MHD !!!

#  elif ( MODEL != ELBDM )
#  warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#  endif // MODEL


// free the host memory allocated by CUDA
   for (int t=0; t<2; t++)
   {
      if ( h_Flu_Array_F_In       [t] != NULL ) CUDA_CHECK_ERROR(  hipHostFree( h_Flu_Array_F_In       [t] )  );
      if ( h_Flu_Array_F_Out      [t] != NULL ) CUDA_CHECK_ERROR(  hipHostFree( h_Flu_Array_F_Out      [t] )  );
      if ( h_Flux_Array           [t] != NULL ) CUDA_CHECK_ERROR(  hipHostFree( h_Flux_Array           [t] )  );
      if ( h_MinDtInfo_Fluid_Array[t] != NULL ) CUDA_CHECK_ERROR(  hipHostFree( h_MinDtInfo_Fluid_Array[t] )  );
      h_Flu_Array_F_In       [t] = NULL;  
      h_Flu_Array_F_Out      [t] = NULL;
      h_Flux_Array           [t] = NULL;
      h_MinDtInfo_Fluid_Array[t] = NULL;
   }


// destroy streams
   if ( Stream != NULL )
   {
      for (int s=0; s<GPU_NStream; s++)   
      {
         CUDA_CHECK_ERROR(  hipStreamDestroy( Stream[s] )  );
      }

      delete [] Stream;
      Stream = NULL;
   }

} // FUNCTION : CUAPI_MemFree_Fluid



#endif // #ifdef GPU
