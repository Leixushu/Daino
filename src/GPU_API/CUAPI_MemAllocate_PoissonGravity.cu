
#include "DAINO.h"

#if ( defined GPU  &&  defined GRAVITY )



extern real (*d_Rho_Array_P    )[ RHO_NXT*RHO_NXT*RHO_NXT ];
extern real (*d_Pot_Array_P_In )[ POT_NXT*POT_NXT*POT_NXT ];
extern real (*d_Pot_Array_P_Out)[ GRA_NXT*GRA_NXT*GRA_NXT ];
extern real (*d_Flu_Array_G    )[GRA_NIN][ PATCH_SIZE*PATCH_SIZE*PATCH_SIZE ];




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_MemAllocate_PoissonGravity
// Description :  Allocate device and host memory for the Poisson and Gravity solvers
//
// Parameter   :  Pot_NPatchGroup   : Number of patch groups evaluated simultaneously by GPU 
//-------------------------------------------------------------------------------------------------------
void CUAPI_MemAllocate_PoissonGravity( const int Pot_NPatchGroup )
{
   
   const long Pot_NPatch        = 8*Pot_NPatchGroup;
   const long Rho_MemSize_P     = sizeof(real)*Pot_NPatch*RHO_NXT   *RHO_NXT   *RHO_NXT;
   const long Pot_MemSize_P_In  = sizeof(real)*Pot_NPatch*POT_NXT   *POT_NXT   *POT_NXT;
   const long Pot_MemSize_P_Out = sizeof(real)*Pot_NPatch*GRA_NXT   *GRA_NXT   *GRA_NXT;
   const long Flu_MemSize_G     = sizeof(real)*Pot_NPatch*PATCH_SIZE*PATCH_SIZE*PATCH_SIZE*GRA_NIN;


// output the total memory requirement
   long TotalSize = Rho_MemSize_P + Pot_MemSize_P_In + Pot_MemSize_P_Out + Flu_MemSize_G;

   if ( MPI_Rank == 0 )
      Aux_Message( stdout, "NOTE : total memory requirement in GPU Poisson and gravity solver = %ld MB\n", 
                   TotalSize/(1<<20) ); 


// allocate the device memory
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Rho_Array_P,     Rho_MemSize_P     )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Pot_Array_P_In,  Pot_MemSize_P_In  )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Pot_Array_P_Out, Pot_MemSize_P_Out )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Flu_Array_G,     Flu_MemSize_G     )  );


// allocate the host memory by CUDA
   for (int t=0; t<2; t++)
   {
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Rho_Array_P    [t], Rho_MemSize_P     )  );
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Pot_Array_P_In [t], Pot_MemSize_P_In  )  );
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Pot_Array_P_Out[t], Pot_MemSize_P_Out )  );
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Flu_Array_G    [t], Flu_MemSize_G     )  );
   }

} // FUNCTION : CUAPI_MemAllocate_PoissonGravity



#endif // #if ( defined GPU  &&  defined GRAVITY )
