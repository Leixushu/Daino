#include "hip/hip_runtime.h"

#include "DAINO.h"
#include "CUFLU.h"
#ifdef GRAVITY
#include "CUPOT.h"
#endif

#ifdef GPU



// fluid solver prototypes in different models
#if   ( MODEL == HYDRO )
#if   ( FLU_SCHEME == RTVD )
__global__ void CUFLU_FluidSolver_RTVD( real g_Fluid_In [][5][ FLU_NXT*FLU_NXT*FLU_NXT ], 
                                        real g_Fluid_Out[][5][ PS2*PS2*PS2 ],
                                        real g_Flux[][9][5][ PS2*PS2 ], 
                                        const real dt, const real _dh, const real Gamma, const bool StoreFlux,
                                        const bool XYZ );
#elif ( FLU_SCHEME == WAF )
__global__ void CUFLU_FluidSolver_WAF( real g_Fluid_In [][5][ FLU_NXT*FLU_NXT*FLU_NXT ], 
                                       real g_Fluid_Out[][5][ PS2*PS2*PS2 ],
                                       real g_Flux[][9][5][ PS2*PS2 ], 
                                       const real dt, const real _dh, const real Gamma, const bool StoreFlux,
                                       const bool XYZ, const WAF_Limiter_t WAF_Limiter );
#elif ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP )
__global__ void CUFLU_FluidSolver_MHM( const real g_Fluid_In[][5][ FLU_NXT*FLU_NXT*FLU_NXT ], 
                                       real g_Fluid_Out  [][5][ PS2*PS2*PS2 ],
                                       real g_Flux    [][9][5][ PS2*PS2 ], 
                                       real g_PriVar     [][5][ FLU_NXT*FLU_NXT*FLU_NXT ],
                                       real g_Slope_PPM_x[][5][ N_SLOPE_PPM*N_SLOPE_PPM*N_SLOPE_PPM],
                                       real g_Slope_PPM_y[][5][ N_SLOPE_PPM*N_SLOPE_PPM*N_SLOPE_PPM],
                                       real g_Slope_PPM_z[][5][ N_SLOPE_PPM*N_SLOPE_PPM*N_SLOPE_PPM],
                                       real g_FC_Var_xL  [][5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ], 
                                       real g_FC_Var_xR  [][5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ], 
                                       real g_FC_Var_yL  [][5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ], 
                                       real g_FC_Var_yR  [][5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ],
                                       real g_FC_Var_zL  [][5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ], 
                                       real g_FC_Var_zR  [][5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ],
                                       real g_FC_Flux_x  [][5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ],
                                       real g_FC_Flux_y  [][5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ],
                                       real g_FC_Flux_z  [][5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ],
                                       const real dt, const real _dh, const real Gamma, const bool StoreFlux,
                                       const LR_Limiter_t LR_Limiter, const real MinMod_Coeff, 
                                       const real EP_Coeff );
#elif ( FLU_SCHEME == CTU )
__global__ void CUFLU_FluidSolver_CTU( const real g_Fluid_In[][5][ FLU_NXT*FLU_NXT*FLU_NXT ], 
                                       real g_Fluid_Out  [][5][ PS2*PS2*PS2 ],
                                       real g_Flux    [][9][5][ PS2*PS2 ], 
                                       real g_PriVar     [][5][ FLU_NXT*FLU_NXT*FLU_NXT ],
                                       real g_Slope_PPM_x[][5][ N_SLOPE_PPM*N_SLOPE_PPM*N_SLOPE_PPM],
                                       real g_Slope_PPM_y[][5][ N_SLOPE_PPM*N_SLOPE_PPM*N_SLOPE_PPM],
                                       real g_Slope_PPM_z[][5][ N_SLOPE_PPM*N_SLOPE_PPM*N_SLOPE_PPM],
                                       real g_FC_Var_xL  [][5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ], 
                                       real g_FC_Var_xR  [][5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ], 
                                       real g_FC_Var_yL  [][5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ], 
                                       real g_FC_Var_yR  [][5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ],
                                       real g_FC_Var_zL  [][5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ], 
                                       real g_FC_Var_zR  [][5][ N_FC_VAR*N_FC_VAR*N_FC_VAR ],
                                       real g_FC_Flux_x  [][5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ],
                                       real g_FC_Flux_y  [][5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ],
                                       real g_FC_Flux_z  [][5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ],
                                       const real dt, const real _dh, const real Gamma, const bool StoreFlux,
                                       const LR_Limiter_t LR_Limiter, const real MinMod_Coeff, 
                                       const real EP_Coeff );
#endif
//__global__ void CUFLU_GetMaxCFL( real g_Fluid[][5][ PS2*PS2*PS2 ], real g_MaxCFL[], const real Gamma );

#elif ( MODEL == MHD )
#warning :: WAIT MHD !!!

#elif ( MODEL == ELBDM )
__global__ void CUFLU_ELBDMSolver( real g_Fluid_In [][FLU_NIN ][ FLU_NXT*FLU_NXT*FLU_NXT ], 
                                   real g_Fluid_Out[][FLU_NOUT][ PS2*PS2*PS2 ],
                                   const real dt, const real _dh, const real Eta, const bool XYZ );

#else
#error : ERROR : unsupported MODEL !!
#endif // MODEL


#ifdef GRAVITY

// Poisson solver prototypes
#if   ( POT_SCHEME == SOR )
#ifdef USE_PSOLVER_10TO14
__global__ void CUPOT_PoissonSolver_SOR_10to14cube( const real g_Rho_Array    [][ RHO_NXT*RHO_NXT*RHO_NXT ], 
                                                    const real g_Pot_Array_In [][ POT_NXT*POT_NXT*POT_NXT ], 
                                                          real g_Pot_Array_Out[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                                    const int Min_Iter, const int Max_Iter, const real Omega_6,
                                                    const real Const, const IntScheme_t IntScheme );
#else
__global__ void CUPOT_PoissonSolver_SOR_16to18cube( const real g_Rho_Array    [][ RHO_NXT*RHO_NXT*RHO_NXT ], 
                                                    const real g_Pot_Array_In [][ POT_NXT*POT_NXT*POT_NXT ], 
                                                          real g_Pot_Array_Out[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                                    const int Min_Iter, const int Max_Iter, const real Omega_6, 
                                                    const real Const, const IntScheme_t IntScheme );
#endif // #ifdef USE_PSOLVER_10TO14 ... else ...
#elif ( POT_SCHEME == MG )
__global__ void CUPOT_PoissonSolver_MG( const real g_Rho_Array    [][ RHO_NXT*RHO_NXT*RHO_NXT ], 
                                        const real g_Pot_Array_In [][ POT_NXT*POT_NXT*POT_NXT ], 
                                              real g_Pot_Array_Out[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                        const real dh_Min, const int Max_Iter, const int NPre_Smooth,
                                        const int NPost_Smooth, const real Tolerated_Error, const real Poi_Coeff,
                                        const IntScheme_t IntScheme );
#endif // POT_SCHEME


// Gravity solver prototypes in different models
#if   ( MODEL == HYDRO )
__global__ void CUPOT_HydroGravitySolver(       real g_Flu_Array[][5][ PATCH_SIZE*PATCH_SIZE*PATCH_SIZE ],
                                          const real g_Pot_Array[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                          const real Gra_Const, const bool P5_Gradient );
#elif ( MODEL == MHD )
#warning :: WAIT MHD !!!

#elif ( MODEL == ELBDM )
__global__ void CUPOT_ELBDMGravitySolver(       real g_Flu_Array[][GRA_NIN][ PATCH_SIZE*PATCH_SIZE*PATCH_SIZE ],
                                          const real g_Pot_Array[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                          const real EtaDt );

#else
#error : ERROR : unsupported MODEL !!
#endif // MODEL

#endif // GRAVITY




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_Set_Default_GPU_Parameter
// Description :  Set several GPU parameters to the default values if they are not set in the input file
//
// Parameter   :  GPU_NStream       : Number of streams for the asynchronous memory copy in GPU 
//                Flu_GPU_NPGroup   : Number of patch groups sent into GPU simultaneously for the fluid solver
//                Pot_GPU_NPGroup   : Number of patch groups sent into GPU simultaneously for the Poisson solver
//-------------------------------------------------------------------------------------------------------
void CUAPI_Set_Default_GPU_Parameter( int &GPU_NStream, int &Flu_GPU_NPGroup, int &Pot_GPU_NPGroup )
{

   if ( MPI_Rank == 0 )    Aux_Message( stdout, "%s ... \n", __FUNCTION__ );


// get the device ID
   int GetDeviceID = 999;
   CUDA_CHECK_ERROR(  hipGetDevice( &GetDeviceID )  );


// load the device properties
   hipDeviceProp_t DeviceProp;
   CUDA_CHECK_ERROR(  hipGetDeviceProperties( &DeviceProp, GetDeviceID )  );


// set the default GPU parameters
// (1) GPU_NSTREAM  
   if ( GPU_NStream < 0 )
   {
      if ( DeviceProp.deviceOverlap )     
      {
#        if   ( MODEL == HYDRO )
         GPU_NStream = 8;

#        elif ( MODEL == MHD )
#        warning :: WAIT MHD !!!

#        elif ( MODEL == ELBDM )
//       GPU_NStream = 16;
         GPU_NStream = 8;

#        else
#        error : ERROR : unsupported MODEL !!
#        endif // MODEL
      }
      else                                
         GPU_NStream = 1;

      if ( MPI_Rank == 0 )  Aux_Message( stdout, "NOTE : parameter \"%s\" is set to the default value = %d\n",
                                         "GPU_NSTREAM", GPU_NSTREAM );
   }

// (2) FLU_GPU_NPGROUP
   if ( Flu_GPU_NPGroup < 0 )  
   {
#     if   ( MODEL == HYDRO )
#        ifdef FERMI
//       Flu_GPU_NPGroup = 4*GPU_NStream*DeviceProp.multiProcessorCount;
//       Flu_GPU_NPGroup = 2*GPU_NStream*DeviceProp.multiProcessorCount;
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        else
         Flu_GPU_NPGroup = 2*GPU_NStream*DeviceProp.multiProcessorCount;
#        endif

#     elif ( MODEL == MHD )
#        warning :: WAIT MHD !!!

#     elif ( MODEL == ELBDM )
#        ifdef FERMI
//       Flu_GPU_NPGroup = 4*GPU_NStream*DeviceProp.multiProcessorCount;
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        else
//       Flu_GPU_NPGroup = 2*GPU_NStream*DeviceProp.multiProcessorCount;
         Flu_GPU_NPGroup = 1*GPU_NStream*DeviceProp.multiProcessorCount;
#        endif

#     else
#        error : ERROR : unsupported MODEL !!
#     endif // MODEL

      if ( MPI_Rank == 0 )  Aux_Message( stdout, "NOTE : parameter \"%s\" is set to the default value = %d\n",
                                         "FLU_GPU_NPGROUP", Flu_GPU_NPGroup );
   }

// (3) POT_GPU_NPGROUP
#  ifdef GRAVITY
   if ( Pot_GPU_NPGroup < 0 )  
   {
#     ifdef FERMI
//    Pot_GPU_NPGroup = 4*DeviceProp.multiProcessorCount*GPU_NStream;
//    Pot_GPU_NPGroup = 2*DeviceProp.multiProcessorCount*GPU_NStream;
      Pot_GPU_NPGroup = 1*DeviceProp.multiProcessorCount*GPU_NStream;
#     else
      Pot_GPU_NPGroup = 2*DeviceProp.multiProcessorCount*GPU_NStream;
#     endif

      if ( MPI_Rank == 0 )  Aux_Message( stdout, "NOTE : parameter \"%s\" is set to the default value = %d\n",
                                         "POT_GPU_NPGROUP", Pot_GPU_NPGroup );
   }
#  endif

// (4) cache preference for Fermi GPUs
#  ifdef FERMI

//    (4-1) fluid solver
#     if   ( MODEL == HYDRO )
#     if   ( FLU_SCHEME == RTVD )   
      CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_FluidSolver_RTVD), hipFuncCachePreferShared )  );
#     elif ( FLU_SCHEME == WAF )
      CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_FluidSolver_WAF),  hipFuncCachePreferShared )  );
#     elif ( FLU_SCHEME == MHM )
      CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_FluidSolver_MHM),  hipFuncCachePreferL1     )  );
#     elif ( FLU_SCHEME == MHM_RP )
      CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_FluidSolver_MHM),  hipFuncCachePreferL1     )  );
#     elif ( FLU_SCHEME == CTU )
      CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_FluidSolver_CTU),  hipFuncCachePreferL1     )  );
#     endif

//    CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_GetMaxCFL),        hipFuncCachePreferShared )  );

#     elif ( MODEL == MHD )
#     warning :: WAIT MHD !!!

#     elif ( MODEL == ELBDM )
      CUDA_CHECK_ERROR(  hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUFLU_ELBDMSolver),      hipFuncCachePreferShared )  );

#     else
#     error : ERROR : unsupported MODEL !!
#     endif // MODEL


#     ifdef GRAVITY      

//    (4-2) Poisson solver
#     if   ( POT_SCHEME == SOR )
#     ifdef USE_PSOLVER_10TO14
      CUDA_CHECK_ERROR( hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_PoissonSolver_SOR_10to14cube), hipFuncCachePreferShared ) );
#     else
      CUDA_CHECK_ERROR( hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_PoissonSolver_SOR_16to18cube), hipFuncCachePreferShared ) );
#     endif
#     elif ( POT_SCHEME == MG )
      CUDA_CHECK_ERROR( hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_PoissonSolver_MG),             hipFuncCachePreferShared ) );
#     endif // POT_SCHEME


//    (4-3) gravity solver
#     if   ( MODEL == HYDRO )
      CUDA_CHECK_ERROR( hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_HydroGravitySolver),           hipFuncCachePreferShared ) );

#     elif ( MODEL == MHD )
#     warning : WAIT MHD !!!

#     elif ( MODEL == ELBDM )
      CUDA_CHECK_ERROR( hipFuncSetCacheConfig(reinterpret_cast<const void*>( CUPOT_ELBDMGravitySolver),           hipFuncCachePreferL1     ) );

#     else
#     error : ERROR : unsupported MODEL !!
#     endif // MODEL

#     endif // GRAVITY

#  endif // FERMMI


// (5) set the constant variables
#  ifdef GRAVITY      
   const real Mp[3] = { -3.0/32.0, +30.0/32.0, +5.0/32.0 };
   const real Mm[3] = { +5.0/32.0, +30.0/32.0, -3.0/32.0 };

   hipMemcpyToSymbol(HIP_SYMBOL( "Mp"), Mp, sizeof(Mp) );
   hipMemcpyToSymbol(HIP_SYMBOL( "Mm"), Mm, sizeof(Mm) );
#  endif


   if ( MPI_Rank == 0 )    Aux_Message( stdout, "%s ... done\n", __FUNCTION__ );

} // FUNCTION : CUAPI_Set_Default_GPU_Parameter



#endif // #ifdef GPU
