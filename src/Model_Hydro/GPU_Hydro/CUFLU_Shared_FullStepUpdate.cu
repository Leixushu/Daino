#include "hip/hip_runtime.h"

#ifndef __CUFLU_FULLSTEPUPDATE_CU__
#define __CUFLU_FULLSTEPUPDATE_CU__



#include "Macro.h"
#include "CUFLU.h"

static __device__ void CUFLU_FullStepUpdate( const real g_Fluid_In[][5][ FLU_NXT*FLU_NXT*FLU_NXT ], 
                                             real g_Fluid_Out[][5][ PS2*PS2*PS2 ],
                                             const real g_FC_Flux_x[][5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ],
                                             const real g_FC_Flux_y[][5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ],
                                             const real g_FC_Flux_z[][5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ],
                                             const real dt, const real _dh, const real Gamma );




//-------------------------------------------------------------------------------------------------------
// Function    :  CUFLU_FullStepUpdate
// Description :  Evaluate the full-step solution 
//
// Note        :  1. Prefix "g" for pointers pointing to the "Global" memory space
//                   Prefix "s" for pointers pointing to the "Shared" memory space
//                2. This function is shared by MHM, MHM_RP, and CTU schemes
//
// Parameter   :  g_Fluid_In  : Global memory array storing the input fluid variables
//                g_Fluid_Out : Global memory array to store the output fluid variables
//                g_FC_Flux_x : Global memory array storing the input face-centered fluxes in the x direction
//                g_FC_Flux_y : Global memory array storing the input face-centered fluxes in the y direction
//                g_FC_Flux_z : Global memory array storing the input face-centered fluxes in the z direction
//                dt          : Time interval to advance solution
//                _dh         : 1 / grid size
//                Gamma       : Ratio of specific heats
//-------------------------------------------------------------------------------------------------------
__device__ void CUFLU_FullStepUpdate( const real g_Fluid_In [][5][ FLU_NXT*FLU_NXT*FLU_NXT ], 
                                            real g_Fluid_Out[][5][ PS2*PS2*PS2 ],
                                      const real g_FC_Flux_x[][5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ],
                                      const real g_FC_Flux_y[][5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ],
                                      const real g_FC_Flux_z[][5][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ],
                                      const real dt, const real _dh, const real Gamma )
{

   const uint  bx      = blockIdx.x;
   const uint  tx      = threadIdx.x;
   const uint  dID_Out = blockDim.x;
   const uint3 dID_F   = make_uint3( 1, N_FL_FLUX, N_FL_FLUX*N_FL_FLUX );
   const real  dt_dh   = dt*_dh;

   uint   ID_In, ID_F, ID_Out;
   uint3  ID3d;
   FluVar ConVar;
   real   FluxDiff;

#  ifdef ENFORCE_POSITIVE
   const real  Gamma_m1 = Gamma - (real)1.0;
   const real _Gamma_m1 = (real)1.0 / Gamma_m1;
   real   Ek, TempPres;
#  endif


   ID_Out = tx;

// loop over all cells
   while ( ID_Out < PS2*PS2*PS2 )
   {
      ID3d.x = ID_Out%PS2;
      ID3d.y = ID_Out%(PS2*PS2)/PS2;
      ID3d.z = ID_Out/(PS2*PS2);
      ID_In  = __umul24( __umul24( ID3d.z+FLU_GHOST_SIZE, FLU_NXT  ) + ID3d.y+FLU_GHOST_SIZE, FLU_NXT  )
               + ID3d.x+FLU_GHOST_SIZE;
      ID_F   = __umul24( __umul24( ID3d.z, N_FL_FLUX ) + ID3d.y, N_FL_FLUX ) + ID3d.x;


//    get the full-step solution
      ConVar.Rho = g_Fluid_In[bx][0][ID_In];
      ConVar.Px  = g_Fluid_In[bx][1][ID_In];
      ConVar.Py  = g_Fluid_In[bx][2][ID_In];
      ConVar.Pz  = g_Fluid_In[bx][3][ID_In];
      ConVar.Egy = g_Fluid_In[bx][4][ID_In];

#     define Update( comp, v )                                                                        \
      {                                                                                               \
         FluxDiff = dt_dh * (  g_FC_Flux_x[bx][v][ID_F+dID_F.x] - g_FC_Flux_x[bx][v][ID_F] +          \
                               g_FC_Flux_y[bx][v][ID_F+dID_F.y] - g_FC_Flux_y[bx][v][ID_F] +          \
                               g_FC_Flux_z[bx][v][ID_F+dID_F.z] - g_FC_Flux_z[bx][v][ID_F]  );        \
         ConVar.comp -= FluxDiff;                                                                     \
      } // Update

      Update( Rho, 0 );
      Update( Px,  1 );
      Update( Py,  2 );
      Update( Pz,  3 );
      Update( Egy, 4 );

#     undef Update


//    enforce the pressure to be positive
#     ifdef ENFORCE_POSITIVE
      Ek         = (real)0.5*( ConVar.Px*ConVar.Px + ConVar.Py*ConVar.Py + ConVar.Pz*ConVar.Pz )/ConVar.Rho;     
      TempPres   = Gamma_m1*( ConVar.Egy - Ek );                                               
      TempPres   = FMAX( TempPres, MIN_VALUE );                                            
      ConVar.Egy = Ek + _Gamma_m1*TempPres;                                                
#     endif


//    save the updated data back to the output global array
      g_Fluid_Out[bx][0][ID_Out] = ConVar.Rho;
      g_Fluid_Out[bx][1][ID_Out] = ConVar.Px;
      g_Fluid_Out[bx][2][ID_Out] = ConVar.Py;
      g_Fluid_Out[bx][3][ID_Out] = ConVar.Pz;
      g_Fluid_Out[bx][4][ID_Out] = ConVar.Egy;


      ID_Out += dID_Out;

   } // while ( ID_Out < PS2*PS2*PS2 )

} // FUNCTION : CUFLU_FullStepUpdate



#endif // #ifndef __CUFLU_FULLSTEPUPDATE_CU__
