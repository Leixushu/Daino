#include "hip/hip_runtime.h"

#include "Macro.h"
#include "CUFLU.h"

#if ( defined GPU  &&  MODEL == HYDRO )



//-------------------------------------------------------------------------------------------------------
// Function    :  CUFLU_GetMaxCFL
// Description :  Evaluate the maximum propagation speed in each patch group 
//
// Note        :  Prefix "g" for pointers pointing to the "Global" memory space
//                Prefix "s" for pointers pointing to the "Shared" memory space
//
// Parameter   :  g_Fluid  : Global memory array to store the fluid variables
//                g_MaxCFL : Global memory array to store the maximum propagation speed in each patch group
//                Gamma    : Ratio of specific heats
//-------------------------------------------------------------------------------------------------------
__global__ void CUFLU_GetMaxCFL( real g_Fluid[][5][ PS2*PS2*PS2 ], real g_MaxCFL[], const real Gamma )
{

   const uint bx       = blockIdx.x;
   const uint tx       = threadIdx.x; 
   const uint ty       = threadIdx.y; 
   const uint ID0      = ty*PS2 + tx;
   const real Gamma_m1 = Gamma - (real)1.0;

   real u[5], Ek, Pres, Cs, MaxV;
   int ID;

   volatile __shared__ real s_MaxCFL_xy[PS2*PS2];
   volatile __shared__ real s_MaxCFL_z [PS2];


   for (int z=0; z<PS2; z++)
   {
      ID = ID0 + z*PS2*PS2;

      u[0] = (real)1.0 / g_Fluid[bx][0][ID];
      u[1] = FABS( g_Fluid[bx][1][ID] );
      u[2] = FABS( g_Fluid[bx][2][ID] );
      u[3] = FABS( g_Fluid[bx][3][ID] );
      u[4] = g_Fluid[bx][4][ID];

      Ek   = (real)0.5*( u[1]*u[1] + u[2]*u[2] + u[3]*u[3] )*u[0]; 
      Pres = Gamma_m1*( u[4] - Ek );
#     ifdef ENFORCE_POSITIVE
//    Pres = FMAX( Pres, MIN_VALUE ); // replaced by the following line for detecting NaN
      Pres = ( Pres < MIN_VALUE ) ? MIN_VALUE : Pres;
#     endif
      Cs   = SQRT( Gamma*Pres*u[0] );

#     if   ( FLU_SCHEME == RTVD  ||  FLU_SCHEME == CTU  ||  FLU_SCHEME == WAF )
      MaxV             = ( u[1] > u[2] ) ? u[1] : u[2];
      MaxV             = ( u[3] > MaxV ) ? u[3] : MaxV;
      MaxV            *= u[0];
      s_MaxCFL_xy[ID0] = MaxV + Cs;

#     elif ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP )
      MaxV             = u[0]*( u[1] + u[2] + u[3] );
      s_MaxCFL_xy[ID0] = MaxV + (real)3.0*Cs;
#     endif

      __syncthreads();


//    perform the reduction operation to get the maximum CFL speed in each z slice
      if ( ID0 < 128 )
         s_MaxCFL_xy[ID0] = ( s_MaxCFL_xy[ID0+128] > s_MaxCFL_xy[ID0] ) ? s_MaxCFL_xy[ID0+128] : s_MaxCFL_xy[ID0];

      __syncthreads();

      if ( ID0 < 64 )
         s_MaxCFL_xy[ID0] = ( s_MaxCFL_xy[ID0+ 64] > s_MaxCFL_xy[ID0] ) ? s_MaxCFL_xy[ID0+ 64] : s_MaxCFL_xy[ID0];

      __syncthreads();

      if ( ID0 < 32 ) 
      {  
         s_MaxCFL_xy[ID0] = ( s_MaxCFL_xy[ID0+ 32] > s_MaxCFL_xy[ID0] ) ? s_MaxCFL_xy[ID0+ 32] : s_MaxCFL_xy[ID0];
         s_MaxCFL_xy[ID0] = ( s_MaxCFL_xy[ID0+ 16] > s_MaxCFL_xy[ID0] ) ? s_MaxCFL_xy[ID0+ 16] : s_MaxCFL_xy[ID0];
         s_MaxCFL_xy[ID0] = ( s_MaxCFL_xy[ID0+  8] > s_MaxCFL_xy[ID0] ) ? s_MaxCFL_xy[ID0+  8] : s_MaxCFL_xy[ID0];
         s_MaxCFL_xy[ID0] = ( s_MaxCFL_xy[ID0+  4] > s_MaxCFL_xy[ID0] ) ? s_MaxCFL_xy[ID0+  4] : s_MaxCFL_xy[ID0];
         s_MaxCFL_xy[ID0] = ( s_MaxCFL_xy[ID0+  2] > s_MaxCFL_xy[ID0] ) ? s_MaxCFL_xy[ID0+  2] : s_MaxCFL_xy[ID0];
         s_MaxCFL_xy[ID0] = ( s_MaxCFL_xy[ID0+  1] > s_MaxCFL_xy[ID0] ) ? s_MaxCFL_xy[ID0+  1] : s_MaxCFL_xy[ID0];
      }

      __syncthreads();

      if ( ID0 == 0 )      s_MaxCFL_z[z] = s_MaxCFL_xy[0];

      __syncthreads();

   } // for (int z=FLU_GHOST_SIZE; z<FLU_GHOST_SIZE+PS2; z++)


// perform the reduction operation to get the maximum CFL speed of each patch
   if ( ID0 < 8 ) 
   {  
      s_MaxCFL_z[ID0] = ( s_MaxCFL_z[ID0+8] > s_MaxCFL_z[ID0] ) ? s_MaxCFL_z[ID0+8] : s_MaxCFL_z[ID0];
      s_MaxCFL_z[ID0] = ( s_MaxCFL_z[ID0+4] > s_MaxCFL_z[ID0] ) ? s_MaxCFL_z[ID0+4] : s_MaxCFL_z[ID0];
      s_MaxCFL_z[ID0] = ( s_MaxCFL_z[ID0+2] > s_MaxCFL_z[ID0] ) ? s_MaxCFL_z[ID0+2] : s_MaxCFL_z[ID0];
      s_MaxCFL_z[ID0] = ( s_MaxCFL_z[ID0+1] > s_MaxCFL_z[ID0] ) ? s_MaxCFL_z[ID0+1] : s_MaxCFL_z[ID0];
   }


// store the maximum CFL speed among each patch back to the global memory
   if ( ID0 == 0 )   g_MaxCFL[bx] = s_MaxCFL_z[0];

} // FUNCTION : CUFLU_GetMaxCFL



#endif // #if ( defined GPU  &&  MODEL == HYDRO )
